/**
 * jrc_cuda_rho.cu
 * block loading rho calculation. should be much faster
 * system('nvcc -ptx -m 64 -arch sm_35 jrc_cuda_rho.cu')
 * i1 is multiple of chunk (16)
 * J. James Jun, Vidrio Technologies, LLC., 2017 Jun 11
 * 7/13/17: fDc_spk option added, which uses spike-specific distance cut-off (dc)
*/

#include <hip/hip_runtime.h>
// #include "cublas_v2.h"
#include <math.h>
#define ABS(my_val) ((my_val) < 0) ? (-1*(my_val)) : (my_val)
#define MIN(A,B) ((A)<(B)) ? (A) : (B)
#define MAX(A,B) ((A)>(B)) ? (A) : (B)
#define NTHREADS 128
#define NC 45 //max dimm
#define CHUNK 16
#define SINGLE_INF (3.402E+38) // equipvalent to NAN. consider -1 value

/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 * Step through one B at a time
 */
__global__ void jrc_cuda_rho(float * vrRho1, const float * mrFet12, const int * viiSpk12_ord, const int *  vnConst, const float dc2){
//__global__ void jrc_cuda_rho(int *vnRho1, int *vnComp1, float const *mrFet12, int const *viiSpk12_ord, int const *vnC4, float const dc2){
    int i1 = (blockIdx.x + blockIdx.y * gridDim.x) * CHUNK;   // base index of i1    
    int tx = threadIdx.x; //nThreads for i12 index    
    int i1_tx = i1+tx;
    int n1 = vnConst[0];
    int n12 = vnConst[1];
    int nC = vnConst[2];
    int dn_max = vnConst[3];    
    int fDc_spk = vnConst[4];
    
    __shared__ int viiSpk1_ord_[CHUNK];
    __shared__ float mrFet1_[NC][CHUNK];    
    __shared__ int mnRho1_[NTHREADS][CHUNK]; // count then divide later    
    __shared__ int mnComp1_[NTHREADS][CHUNK]; // count number of elements compared        
    __shared__ float vrDc1_[CHUNK];  // use if fDc_spk=1
    
    // cache shared memory
    if (tx < nC){ //use tx as iC
        for (int i_c = 0; i_c < CHUNK; ++i_c){
            int i1_c = i_c + i1;
            if (i1_c < n1){
                mrFet1_[tx][i_c] = mrFet12[tx + i1_c * nC];
            }else{
                mrFet1_[tx][i_c] = 0.0f;
            }
        }
    }
    if (tx < CHUNK && i1_tx < n1) viiSpk1_ord_[tx] = viiSpk12_ord[i1_tx];
    
    for (int i_c = 0; i_c < CHUNK; ++i_c){
        mnRho1_[tx][i_c] = 0; // initialize rho
        mnComp1_[tx][i_c] = 0;
    }
    
    // calculate spike-specific distance cut-off vrDc1_ only if fDc_spk==1
    if (tx < CHUNK && fDc_spk==1){
        vrDc1_[tx] = 0.0f; //init
        //for (int iC = 0; iC < 1; ++iC){ //center only scale
        for (int iC = 0; iC < nC; ++iC){
            float temp_ = mrFet1_[iC][tx];
            vrDc1_[tx] += (temp_ * temp_);
        }
        vrDc1_[tx] *= dc2;
    }

    __syncthreads();        

    
    // Inspect distance relationship between i1 and i12_tx
    for (int i12_tx = tx; i12_tx < n12; i12_tx += blockDim.x){
    //for (int i12_tx = 1; i12_tx < n12; ++i12_tx){
        // compute time difference
        //char vlDist_c[CHUNK];
        int iiSpk12_ord_tx = viiSpk12_ord[i12_tx];        
        /*for (int i_c = 0; i_c < CHUNK; ++i_c){
            int di_spk_tx = ABS(viiSpk1_ord_[i_c] - iiSpk12_ord_tx);
            vlDist_c[i_c] = (di_spk_tx <= dn_max);
        } */
        
        // compute distance
        float vrDist_c[CHUNK];
        for (int i_c = 0; i_c < CHUNK; ++i_c) vrDist_c[i_c] = 0.0f;        
        for (int iC = 0; iC < nC; ++iC){
            float fet12_tx = mrFet12[iC + i12_tx * nC];
            for (int i_c = 0; i_c < CHUNK; ++i_c){
                float temp = fet12_tx - mrFet1_[iC][i_c];
                vrDist_c[i_c] += temp * temp;
            }            
        }
        
        // Compare the index and distance
        for (int i_c = 0; i_c < CHUNK; ++i_c){
            int di_spk_tx = ABS(viiSpk1_ord_[i_c] - iiSpk12_ord_tx);
            if (di_spk_tx <= dn_max){
            //if (vlDist_c[i_c] == 1){
                ++mnComp1_[tx][i_c];
                if (fDc_spk==0){
                    if (vrDist_c[i_c] <= dc2) ++mnRho1_[tx][i_c];
                }else{
                    if (vrDist_c[i_c] < vrDc1_[i_c]) ++mnRho1_[tx][i_c];
                }
            }
        }
    } // while
    
    // final count
    __syncthreads();
    //if (tx < CHUNK && i1_tx < n1){  // use tx as i_c
    if (tx < CHUNK){  // use tx as i_c
        int nRho1 = 0;
        int nComp1 = 0;
        for (int tx1=0; tx1<blockDim.x; ++tx1){
            nRho1 += mnRho1_[tx1][tx];
            nComp1 += mnComp1_[tx1][tx];
        }
        if (i1_tx < n1){
            //if (nRho1<1) nRho1 = 1; 
            vrRho1[i1_tx] = (float)(((double)(nRho1)) / ((double)nComp1));
        }
        // vnRho1[i1 + i_c_] = nRho1 - 1;
        // vnComp1[i1 + i_c_] = nComp1;
    }
    //vnRho1[0] = blockDim.x; //debug
    //vnComp1[0] = blockDim.x; //debug
} // func