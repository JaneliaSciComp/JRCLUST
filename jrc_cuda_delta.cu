/**
 * jrc_cuda_delta.cu
 * block loading delta calculation. should be much faster
 * system('nvcc -ptx -m 64 -arch sm_35 jrc_cuda_rho.cu')
 * iA is multiple of CHUNK (16)
 * J. James Jun, Vidrio Technologies, LLC., 2017 Jun 11
*/

#include <hip/hip_runtime.h>
// #include "cublas_v2.h"
#include <math.h>
#define ABS(my_val) ((my_val) < 0) ? (-1*(my_val)) : (my_val)
#define MIN(A,B) ((A)<(B)) ? (A) : (B)
#define MAX(A,B) ((A)>(B)) ? (A) : (B)
#define NTHREADS 128
#define NC 45 // number of Channels
#define CHUNK 16 //previously defined as CHUNK
#define SINGLE_INF (3.402E+38)

/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 * Step through one B at a time
 * 7/13/17: fDc_spk option added, which uses spike-specific distance cut-off (dc)
 */

// % Matlab syntax
// mrDist12_ = eucl2_dist_(mrFet12, mrFet12(:,1:n1));  %not sqrt
// mlRemove12_ = bsxfun(@ge, viiRho12_ord, viiRho12_ord(1:n1)') ...
//     | abs(bsxfun(@minus, viiSpk12_ord_, viiSpk12_ord_(1:n1)')) > dn_max;
// mrDist12_(mlRemove12_) = nan;
// [vrDelta1, viNneigh1] = min(mrDist12_);

__global__ void jrc_cuda_delta(float * vrDelta1, unsigned int * viNneigh1, const float * mrFet12, const int * viiSpk12_ord, const int * viiRho12_ord, const int * vnConst, const float dc2){
    // int iA = blockIdx.x * CHUNK;    
    int i1 = (blockIdx.x + blockIdx.y * gridDim.x) * CHUNK;   // base index of i1
    int tx = threadIdx.x;
    int i1_tx = i1 + tx;
    int n1 = vnConst[0];
    int n12 = vnConst[1];
    int nC = vnConst[2];
    int dn_max = vnConst[3];    
    int fDc_spk = vnConst[4];
    
    __shared__ int viiSpk1_ord_[CHUNK];
    __shared__ int viiRho1_ord_[CHUNK];
    __shared__ float mrFet1_[NC][CHUNK];
    __shared__ float mrDelta1_[NTHREADS][CHUNK];
    __shared__ unsigned int miNneigh1_[NTHREADS][CHUNK]; 
    __shared__ float vrDc1_[CHUNK];  // use if fDc_spk=1
    
    // cache shared memory
    if (tx < nC){ //use tx as iC
        for (int i_c = 0; i_c < CHUNK; ++i_c){
            int i1_c = i_c + i1;
            if (i1_c < n1){
                mrFet1_[tx][i_c] = mrFet12[tx + i1_c * nC];
            }else{
                mrFet1_[tx][i_c] = 0.0f;
            }
        }
    }  
    if (tx < CHUNK && i1_tx < n1){
        viiSpk1_ord_[tx] = viiSpk12_ord[i1_tx];
        viiRho1_ord_[tx] = viiRho12_ord[i1_tx];
    }

    float vr_minDist1[CHUNK];
    unsigned int vi_minIdx1[CHUNK];
    for (int i_c = 0; i_c < CHUNK; ++i_c){
        vr_minDist1[i_c] = SINGLE_INF;
        vi_minIdx1[i_c] = i1 + i_c; // self
    }    
    
    // calculate spike-specific distance cut-off vrDc1_ only if fDc_spk==1
    if (tx < CHUNK && fDc_spk==1){
        vrDc1_[tx] = 0.0f; //init
        //for (int iC = 0; iC < 1; ++iC){ //center only scale
        for (int iC = 0; iC < nC; ++iC){
            float temp_ = mrFet1_[iC][tx];
            vrDc1_[tx] += (temp_ * temp_);
        }
        vrDc1_[tx] *= dc2;
    }
    
    __syncthreads();  
    
    
    // fill in the shared memory A
    for (int i12_tx = tx; i12_tx < n12; i12_tx += blockDim.x){
    //for (int i12_tx = 1; i12_tx < n12; ++i12_tx){
        // compute time difference
        char vlDist_c[CHUNK];
        int iiSpk12_ord_tx = viiSpk12_ord[i12_tx];
        int iiRho12_ord_tx = viiRho12_ord[i12_tx];
        for (int i_c = 0; i_c < CHUNK; ++i_c){
            char di_rho_ = (iiRho12_ord_tx < viiRho1_ord_[i_c]);
            int di_spk_ = ABS(viiSpk1_ord_[i_c] - iiSpk12_ord_tx);
            vlDist_c[i_c] = (di_spk_ <= dn_max) && di_rho_;
        }
        
        // compute distance
        float vrDist_c[CHUNK];
        for (int i_c = 0; i_c < CHUNK; ++i_c) vrDist_c[i_c] = 0.0f;        
        for (int iC = 0; iC < nC; ++iC){
            float fet12_tx = mrFet12[iC + i12_tx * nC];
            for (int i_c = 0; i_c < CHUNK; ++i_c){
                float temp = fet12_tx - mrFet1_[iC][i_c];
                vrDist_c[i_c] += temp * temp;
            }            
        }
        
        // Compare the index and distance
        for (int i_c = 0; i_c < CHUNK; ++i_c){            
            if (vrDist_c[i_c] < vr_minDist1[i_c]){
                if (vlDist_c[i_c] == 1){                
                    vr_minDist1[i_c] = vrDist_c[i_c];
                    vi_minIdx1[i_c] = i12_tx;
                }
            }
        }
    } // while
    
    // collect result from each thread
    for (int i_c = 0; i_c < CHUNK; ++i_c){        
        mrDelta1_[tx][i_c] = vr_minDist1[i_c];
        miNneigh1_[tx][i_c] = vi_minIdx1[i_c];
    }
    __syncthreads();    
    
    // final count    
    //if (tx < CHUNK && i1_tx < n1){
    if (tx < CHUNK){
        float minDist1 = SINGLE_INF;
        unsigned int minIdx1 = i1_tx;
        for (int tx1=0; tx1<blockDim.x; ++tx1){
            if (mrDelta1_[tx1][tx] < minDist1){
                minDist1 = mrDelta1_[tx1][tx];
                minIdx1 = miNneigh1_[tx1][tx];
            }
        }
        //vrDelta1[i1_tx] = sqrtf(minDist1);
        if (i1_tx < n1){
            // vrDelta_ = sqrt(abs(single(vrDelta_) / vrDc2_site(iSite))); %normalize and convert dist
            if (fDc_spk==0){
                vrDelta1[i1_tx] = sqrtf(ABS(minDist1) / dc2); 
            }else{
                vrDelta1[i1_tx] = sqrtf(ABS(minDist1) / vrDc1_[tx]); 
                //vrDelta1[i1_tx] = sqrtf(ABS(minDist1));
            }
            viNneigh1[i1_tx] = minIdx1 + 1; //Matlab index output
        }
    }

} // func